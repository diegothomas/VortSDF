#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <>
#include "cudaType.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#define FAKEINIT = {0}
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#define FAKEINIT
#endif

#define STOP_TRANS 1.0e-8
#define CLIP_ALPHA 60.0
#define BACK_R 1.0f
#define BACK_G 1.0f
#define BACK_B 1.0f 
#define PI 3.141592653589793238462643383279502884197


/** Device functions **/
/** Device functions **/
/** Device functions **/
__device__ const float HUBER_EPS = 1.5f / 255.0f;

__device__ float huber_loss(float3 x) 
{
    float3 absx = abs(x);
    float3 r = mix(x * x * 0.5f, (absx - 0.5f * HUBER_EPS) * HUBER_EPS, greaterThan(abs(x), make_float3(HUBER_EPS, HUBER_EPS, HUBER_EPS)));
    return r.x + r.y + r.z;
}

__device__ float3 huber_grad(float3 x) 
{
    return mix(x, sign(x) * HUBER_EPS, greaterThan(abs(x), make_float3(HUBER_EPS, HUBER_EPS, HUBER_EPS)));
}

__device__ float sdf2Alpha(float sdf, float sdf_prev, float inv_s) 
{
    if (sdf_prev > sdf) {
        double sdf_prev_clamp = fmin(CLIP_ALPHA, fmax(double(sdf_prev * inv_s), -CLIP_ALPHA));
        double sdf_clamp = fmin(CLIP_ALPHA, fmax(double(sdf * inv_s), -CLIP_ALPHA));
        return min(1.0f, __double2float_rn((1.0 + exp(-sdf_prev_clamp)) / (1.0 + exp(-sdf_clamp))));
    }
    return 1.0f;
}

__device__ float4 trace_ray(const float* sdf_seg, const float* color_samples, const int* offsets, float *Wpartial, float inv_s, int n) 
{
    float Tpartial = 1.0f;
    //float Wpartial = 0.0f;
    float3 Cpartial = make_float3(0.0f, 0.0f, 0.0f);
    float3 color = make_float3(0.0f, 0.0f, 0.0f);
    float alpha = 0.0f;

    float previous_sdf = 20.0f;
    float sdf = 20.0f;

    int start = offsets[2 * n];
    int end = offsets[2 * n + 1];
    for (int t = start; t < start + end; t++) {

        previous_sdf = sdf_seg[2*t];
        sdf = sdf_seg[2*t+1];

        color.x = color_samples[3 * t];
        color.y = color_samples[3 * t + 1];
        color.z = color_samples[3 * t + 2];

        alpha = sdf2Alpha(sdf, previous_sdf, inv_s);

        Cpartial = Cpartial + color * (1.0f - alpha) * Tpartial;
        *Wpartial = (*Wpartial) + (1.0f - alpha) * Tpartial;
        Tpartial *= alpha;

        if (Tpartial < STOP_TRANS) {// stop if the transmittance is low
            break;
        }

    }

    // return the total color as well as the final transmittance. The background color will be added after.
    return make_float4(Cpartial, Tpartial);
}

__device__ void backward_no_sdf(float3 Ctotal, float Wtotal, float3 TrueColor, float3 grad_color_diff, float Mask, int n,
                        float* grads_color, float* grads_sdf, const float* sdf_seg, const int *neighbors, const float* weights_seg, const float* color_samples, const int* offsets, const int* cell_ids,
                        float inv_s, float MaskReg = 0.0f, float colorDiscrepancyReg = 0.0f, float BackgroundEntropyReg = 0.0f, int NoColorSpilling = 0) 
{

    float Tpartial = 1.0f;
    float Wpartial = 0.0f;
    float3 Cpartial = make_float3(0.0f, 0.0f, 0.0f);
    float3 color = make_float3(0.0f, 0.0f, 0.0f);
    float3 dc = make_float3(0.0f, 0.0f, 0.0f);
    float3 dCtotal_dalpha = make_float3(0.0f, 0.0f, 0.0f);
    float3 sample_color_diff = make_float3(0.0f, 0.0f, 0.0f);
    float3 A = make_float3(0.0f, 0.0f, 0.0f);

    float sdf_prev = 20.0f; // some large value
    float sdf = 20.0f; // some large value

    int num_knn = 8;

    float alpha, dalpha_dsdf_p, dalpha_dsdf_n, dalpha, contrib;
    double sdf_prev_clamp, sdf_clamp, inv_clipped_p, inv_clipped;
    int id, id_prev;

    int start = offsets[2 * n];
    int end = offsets[2 * n + 1];
    for (int t = start; t < start + end; t++) {        
        
        sdf_prev = sdf_seg[2*t];
        sdf = sdf_seg[2*t+1];

        color.x = color_samples[3 * t];
        color.y = color_samples[3 * t + 1];
        color.z = color_samples[3 * t + 2];

        alpha = 1.0f;
        sdf_prev_clamp = fmin(CLIP_ALPHA, fmax(double(sdf_prev * inv_s), -CLIP_ALPHA));
        sdf_clamp = fmin(CLIP_ALPHA, fmax(double(sdf * inv_s), -CLIP_ALPHA));
        inv_clipped_p = (fabs(sdf_prev) < CLIP_ALPHA / inv_s) ? double(inv_s) : sdf_prev_clamp / double(sdf_prev);
        inv_clipped = (fabs(sdf) < CLIP_ALPHA / inv_s) ? double(inv_s) : sdf_clamp / double(sdf);

        alpha = min(1.0f, __double2float_rn((1.0 + exp(-sdf_prev_clamp)) / (1.0 + exp(-sdf_clamp))));
        if (sdf_prev > sdf && alpha < 1.0f) { // && sdf_prev > 0.0f        
        //if (sdf_prev*sdf <= 0.0f || 
        //        (sdf_prev > sdf && (fabs(sdf)*inv_s < CLIP_ALPHA || fabs(sdf_prev)*inv_s < CLIP_ALPHA))){
            //alpha = min(1.0f, __double2float_rn((1.0 + exp(-sdf_prev_clamp)) / (1.0 + exp(-sdf_clamp))));
            dalpha_dsdf_p = (fabs(sdf_prev * inv_s) > CLIP_ALPHA) ? 0.0f : __double2float_rn(-inv_clipped_p * exp(-sdf_prev_clamp) / (1.0 + exp(-sdf_clamp)));
            //dalpha_dsdf_p = __double2float_rn(-inv_clipped_p * exp(-sdf_prev_clamp) / (1.0 + exp(-sdf_clamp)));
            dalpha_dsdf_n = (fabs(sdf * inv_s) > CLIP_ALPHA) ? 0.0f : __double2float_rn((1.0 + exp(-sdf_prev_clamp)) * ((inv_clipped * exp(-sdf_clamp)) / ((1.0 + exp(-sdf_clamp)) * (1.0 + exp(-sdf_clamp)))));
            //dalpha_dsdf_n = __double2float_rn((1.0 + exp(-sdf_prev_clamp)) * ((inv_clipped * exp(-sdf_clamp)) / ((1.0 + exp(-sdf_clamp)) * (1.0 + exp(-sdf_clamp)))));
        }
        else {
            alpha = 1.0f;
            dalpha_dsdf_p = 0.0f;
            dalpha_dsdf_n = 0.0f;
            continue;
        }
        
        contrib = Tpartial * (1.0f - alpha);
        Cpartial = Cpartial + color * contrib;
        Wpartial = Wpartial + contrib;
        dCtotal_dalpha = ((Ctotal - Cpartial) / alpha) - (color * Tpartial);  // equation 13 from the supplemental
        
        
        ///////////////////////////////////////////////////////// Photometric loss
        dalpha = dot(grad_color_diff, dCtotal_dalpha);
        dc = grad_color_diff * contrib;

        sample_color_diff = Ctotal - color;
        A = huber_grad(sample_color_diff) * contrib;
        
        ///////////////////////////////////////////////////////// Color discrepancy loss
        dalpha += colorDiscrepancyReg * (-huber_loss(sample_color_diff) * Tpartial + dot(A, dCtotal_dalpha));
        dc = dc + A * (contrib - 1.0f) * colorDiscrepancyReg;

        ///////////////////////////////////////////////////////// Background entropy loss
        dalpha += alpha == 0.0f? 0.0f: (1.0f - 2.0f * Wtotal) * (Wtotal / alpha) * BackgroundEntropyReg;

        ///////////////////////////////////////////////////////// Mask regularization
        dalpha += 2.0f * (Wtotal - Mask) * ((Wtotal - Wpartial) / alpha - Tpartial) * MaskReg;

        if (NoColorSpilling != 0) {
            dc = dc * Wtotal * Wtotal;
        }

        float lamda = weights_seg[13*t + 12];
        for (int i = 0; i < 6; i++) {
            id_prev = cell_ids[12 * t + i];
            id = cell_ids[12 * t + 6 + i];

            //atomicAdd(&grads_sdf[id_prev], weights_seg[6*t + i] * dalpha * dalpha_dsdf_p);
            //atomicAdd(&grads_sdf[id], weights_seg[6*t + 3 + i] * dalpha * dalpha_dsdf_n);
            atomicAdd(&grads_color[3 * id_prev], weights_seg[13*t + i] * lamda * dc.x);
            atomicAdd(&grads_color[3 * id_prev+1], weights_seg[13*t + i] * lamda * dc.y);
            atomicAdd(&grads_color[3 * id_prev+2], weights_seg[13*t + i] * lamda * dc.z);
            
            atomicAdd(&grads_color[3 * id], weights_seg[13*t + 6 + i] * (1.0f-lamda) * dc.x);
            atomicAdd(&grads_color[3 * id+1], weights_seg[13*t + 6 + i] * (1.0f-lamda) * dc.y);
            atomicAdd(&grads_color[3 * id+2], weights_seg[13*t + 6 + i] * (1.0f-lamda) * dc.z);
        }

        Tpartial = Tpartial * alpha;

        if (Tpartial < STOP_TRANS) { // stop if the transmittance is low
            break;
        }
    }

    return;
}

__global__ void render_no_sdf_kernel(
    const size_t num_rays,
    const float inv_s,
    const float mask_reg,
    const float *__restrict__ sdf_seg,
    const int *__restrict__ neighbors, 
    const float *__restrict__ weights_seg,
    const float *__restrict__ color_samples,
    const float *__restrict__ true_color,
    const float *__restrict__ mask,
    const int *__restrict__ cell_ids,
    const int *__restrict__ offsets,
    float *__restrict__ grads_sdf,
    float *__restrict__ grads_color,
    float *__restrict__ color_loss,
    float *__restrict__ mask_loss)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rays)
    {
        return;
    }
    
    float Wtotal = 0.0f;
    float4 color = trace_ray(sdf_seg, color_samples, offsets, &Wtotal, inv_s, idx);

    //if (color.w < 1.0f) {
        float msk = mask[idx] > 0.5f ? 1.0f : 0.0f;
        float3 integrated_color = make_float3(color.x + color.w * BACK_R, color.y + color.w * BACK_G, color.z + color.w * BACK_B);
        float3 in_color = make_float3(true_color[3 * idx], true_color[3 * idx + 1], true_color[3 * idx + 2]);
        float3 grad_color_diff = huber_grad(integrated_color - in_color);

        backward_no_sdf(integrated_color, Wtotal, in_color, grad_color_diff, msk,
            idx, grads_color, grads_sdf, sdf_seg, neighbors, weights_seg, color_samples,
            offsets, cell_ids, inv_s, mask_reg, 0.0f, 0.0f, 1);

        //color_loss[3*idx] = integrated_color.x;
        //color_loss[3*idx + 1] = integrated_color.y;
        //color_loss[3*idx + 2] = integrated_color.z;
        color_loss[idx] = msk*huber_loss(integrated_color - in_color);      
        //color_loss[idx] = msk*(fabs(grad_color_diff.x) + fabs(grad_color_diff.y) + fabs(grad_color_diff.z));    
        mask_loss[idx] = (msk - Wtotal)*(msk - Wtotal); //-(msk * logf(Wtotal) + (1.0f - msk) * logf(1.0f-Wtotal));
    //}
    return;
}


__device__ void backward(float3 Ctotal, float Wtotal, float3 TrueColor, float3 grad_color_diff, float Mask, int n,
                        float* grads_color, float* grads_sdf_net, float* grads_sdf, float* counter, const float* sdf_seg, 
                        const int *neighbors, const float* weights_seg, const float* color_samples, const int* offsets, const int* cell_ids,
                        const float *grad_space, const float *rays,
                        float inv_s, float MaskReg = 0.0f, float colorDiscrepancyReg = 0.0f, float BackgroundEntropyReg = 0.0f, int NoColorSpilling = 0) 
{

    float Tpartial = 1.0f;
    float Wpartial = 0.0f;
    float3 Cpartial = make_float3(0.0f, 0.0f, 0.0f);
    float3 color = make_float3(0.0f, 0.0f, 0.0f);
    float3 color_prev = make_float3(0.0f, 0.0f, 0.0f);
    float3 dc = make_float3(0.0f, 0.0f, 0.0f);
    float3 dCtotal_dalpha = make_float3(0.0f, 0.0f, 0.0f);
    float3 sample_color_diff = make_float3(0.0f, 0.0f, 0.0f);
    float3 A = make_float3(0.0f, 0.0f, 0.0f);

    float sdf_prev = 20.0f; // some large value
    float sdf = 20.0f; // some large value

    int num_knn = 8;

    float alpha, dalpha_dsdf_p, dalpha_dsdf_n, dalpha, contrib;
    double sdf_prev_clamp, sdf_clamp, inv_clipped_p, inv_clipped;
    int id, id_prev;

    int start = offsets[2 * n];
    int end = offsets[2 * n + 1];
    for (int t = start; t < start + end; t++) {        
        
        sdf_prev = sdf_seg[2*t];
        sdf = sdf_seg[2*t+1];

        color.x = color_samples[3 * t];
        color.y = color_samples[3 * t + 1];
        color.z = color_samples[3 * t + 2];

        alpha = 1.0f;
        sdf_prev_clamp = fmin(CLIP_ALPHA, fmax(double(sdf_prev * inv_s), -CLIP_ALPHA));
        sdf_clamp = fmin(CLIP_ALPHA, fmax(double(sdf * inv_s), -CLIP_ALPHA));
        inv_clipped_p = (fabs(sdf_prev) < CLIP_ALPHA / inv_s) ? double(inv_s) : sdf_prev_clamp / double(sdf_prev);
        inv_clipped = (fabs(sdf) < CLIP_ALPHA / inv_s) ? double(inv_s) : sdf_clamp / double(sdf);

        alpha = min(1.0f, __double2float_rn((1.0 + exp(-sdf_prev_clamp)) / (1.0 + exp(-sdf_clamp))));
        if (sdf_prev > sdf) { // && alpha < 1.0f) { // && sdf_prev > 0.0f
        //if (sdf_prev*sdf <= 0.0f || 
        //        (sdf_prev > sdf && (fabs(sdf)*inv_s < CLIP_ALPHA || fabs(sdf_prev)*inv_s < CLIP_ALPHA))) {
            //alpha = min(1.0f, __double2float_rn((1.0 + exp(-sdf_prev_clamp)) / (1.0 + exp(-sdf_clamp))));
            //dalpha_dsdf_p = (fabs(sdf_prev * inv_s) > CLIP_ALPHA) ? 0.0f : __double2float_rn(-inv_clipped_p * exp(-sdf_prev_clamp) / (1.0 + exp(-sdf_clamp)));
            dalpha_dsdf_p = __double2float_rn(-inv_clipped_p * exp(-sdf_prev_clamp) / (1.0 + exp(-sdf_clamp)));
            //dalpha_dsdf_n = (fabs(sdf * inv_s) > CLIP_ALPHA) ? 0.0f : __double2float_rn((1.0 + exp(-sdf_prev_clamp)) * ((inv_clipped * exp(-sdf_clamp)) / ((1.0 + exp(-sdf_clamp)) * (1.0 + exp(-sdf_clamp)))));
            dalpha_dsdf_n = __double2float_rn((1.0 + exp(-sdf_prev_clamp)) * ((inv_clipped * exp(-sdf_clamp)) / ((1.0 + exp(-sdf_clamp)) * (1.0 + exp(-sdf_clamp)))));
        }
        else {
            alpha = 1.0f;
            dalpha_dsdf_p = 0.0f;
            dalpha_dsdf_n = 0.0f;
            continue;
        }
        
        /*if (Wtotal > 0.5f) {
            contrib = Tpartial * (1.0f - alpha) / Wtotal;
        } else {
            contrib = Tpartial * (1.0f - alpha);
        }*/
        contrib = Tpartial * (1.0f - alpha);
        Cpartial = Cpartial + color * contrib;
        Wpartial = Wpartial + contrib;
        dCtotal_dalpha = ((Ctotal - Cpartial) / alpha) - (color * Tpartial);  // equation 13 from the supplemental
        
        
        ///////////////////////////////////////////////////////// Photometric loss
        dalpha = dot(grad_color_diff, dCtotal_dalpha);
        dc = grad_color_diff * contrib;

        sample_color_diff = Ctotal - color;
        A = huber_grad(sample_color_diff) * contrib;
        
        ///////////////////////////////////////////////////////// Color discrepancy loss
        //dalpha += colorDiscrepancyReg * (-huber_loss(sample_color_diff) * Tpartial + dot(A, dCtotal_dalpha));
        //dc = dc + A * (contrib - 1.0f) * colorDiscrepancyReg;

        ///////////////////////////////////////////////////////// Background entropy loss
        //dalpha += alpha == 0.0f? 0.0f: (1.0f - 2.0f * Wtotal) * (Wtotal / alpha) * BackgroundEntropyReg;

        ///////////////////////////////////////////////////////// Mask regularization
        /*if (1.0f - Wtotal > 0.5f) {
            //dalpha += 2.0f * (1.0f - Mask) * ((Wtotal - Wpartial) / alpha - Tpartial) * MaskReg;
            dalpha += -2.0f * (1.0f - Mask) * (Wtotal / alpha) * MaskReg;
        } else {
            //dalpha += 2.0f * (Wtotal - Mask) * ((Wtotal - Wpartial) / alpha - Tpartial) * MaskReg;
            dalpha += -2.0f * (1.0f - Wtotal - Mask) * (Wtotal / alpha) * MaskReg;
        }*/
        dalpha += -2.0f * (1.0f - Wtotal - Mask) * (Wtotal / alpha) * MaskReg;

        if (NoColorSpilling != 0) {
            dc = dc * (1.0f - Wtotal) * (1.0f - Wtotal);
        }

        float w_photo = fabs(grad_space[12 * t]*rays[3*n] + grad_space[12 * t + 1]*rays[3*n+1] + grad_space[12 * t + 2]*rays[3*n+2]);
        dalpha = dalpha*w_photo;

        //////////////////////////////////////////////////////////////
        float lambda = weights_seg[7*t + 6] ;
        for (int i = 0; i < 3; i++) {
            id_prev = cell_ids[6 * t + i];
            id = cell_ids[6 * t + 3 + i];
            
			if (lambda < 0.5f) {
                atomicAdd(&grads_sdf[id_prev], weights_seg[7*t + i] * 2.0f*lambda * dalpha * dalpha_dsdf_p);
                atomicAdd(&grads_sdf[id], weights_seg[7*t + 3 + i] * 
                                ((1.0f-2.0f*lambda) * dalpha * dalpha_dsdf_p + dalpha * dalpha_dsdf_n));
                //atomicAdd(&counter[id], 1.0f);
                //atomicAdd(&counter[id_prev], 1.0f);
			} else {
                atomicAdd(&grads_sdf[id_prev], weights_seg[7*t + i] * 
                                    (2.0f*lambda * dalpha * dalpha_dsdf_p + (1.0-2.0f*lambda)*dalpha * dalpha_dsdf_n));
                atomicAdd(&grads_sdf[id], weights_seg[7*t + 3 + i] * (1.0f-(1.0-2.0f*lambda)) * dalpha * dalpha_dsdf_n);
                //atomicAdd(&counter[id], 1.0f);
                //atomicAdd(&counter[id_prev], 1.0f);
			}
        }

        //grads_sdf_net[2 * t] = dalpha * dalpha_dsdf_p;
        //grads_sdf_net[2 * t + 1] = dalpha * dalpha_dsdf_n;

        // add total variation loss along the ray for color
        // float err_tv = 0.5 (color - color_prec)**2

        if (Mask > 0.0f) {
            grads_color[3 * t] = dc.x;
            grads_color[3 * t + 1] = dc.y;
            grads_color[3 * t + 2] = dc.z;
        }

        Tpartial = Tpartial * alpha;

        if (Tpartial < STOP_TRANS) { // stop if the transmittance is low
            break;
        }
    }

    return;
}

__global__ void render_kernel(
    const size_t num_rays,
    const float inv_s,
    const float mask_reg,
    const float *__restrict__ sdf_seg,
    const int *__restrict__ neighbors, 
    const float *__restrict__ weights_seg,
    const float *__restrict__ color_samples,
    const float *__restrict__ true_color,
    const float *__restrict__ mask,
    const int *__restrict__ cell_ids,
    const int *__restrict__ offsets, 
    const float *__restrict__ grad_space, 
    const float *__restrict__ rays,
    float *__restrict__ grads_sdf,
    float *__restrict__ grads_color,
    float *__restrict__ grads_sdf_net,
    float *__restrict__ counter,
    float *__restrict__ color_loss,
    float *__restrict__ mask_loss)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rays)
    {
        return;
    }
    
    float Wtotal = 0.0f;
    float4 color = trace_ray(sdf_seg, color_samples, offsets, &Wtotal, inv_s, idx);

    if (color.w < 1.0f) {
        float msk = mask[idx] > 0.5f ? 1.0f : 0.0f;
        //color.w = color.w > 0.5f ? 1.0f : 0.0f;
        //float3 integrated_color = Wtotal > 0.5f ? make_float3(color.x/Wtotal, color.y/Wtotal, color.z/Wtotal):
        //                            make_float3(color.x + color.w * BACK_R, color.y + color.w * BACK_G, color.z + color.w * BACK_B);
        float3 integrated_color = make_float3(color.x + color.w * BACK_R, color.y + color.w * BACK_G, color.z + color.w * BACK_B);
        float3 in_color = make_float3(true_color[3 * idx], true_color[3 * idx + 1], true_color[3 * idx + 2]);
        float3 grad_color_diff = huber_grad(integrated_color - in_color);

        //Wtotal
        backward(integrated_color, color.w, in_color, grad_color_diff, msk,
            idx, grads_color, grads_sdf_net, grads_sdf, counter, sdf_seg, neighbors, weights_seg, color_samples,
            offsets, cell_ids, grad_space, rays, inv_s, mask_reg, 0.0f, 0.0f, 1);

        //color_loss[3*idx] = integrated_color.x;
        //color_loss[3*idx + 1] = integrated_color.y;
        //color_loss[3*idx + 2] = integrated_color.z;
        color_loss[idx] = msk*huber_loss(integrated_color - in_color);      
        //color_loss[idx] = msk*(fabs(grad_color_diff.x) + fabs(grad_color_diff.y) + fabs(grad_color_diff.z));    
        mask_loss[idx] = (msk - Wtotal)*(msk - Wtotal); //-(msk * logf(Wtotal) + (1.0f - msk) * logf(1.0f-Wtotal));
    }
    return;
}


__global__ void normalize_grads_kernel(
    const size_t num_sites,
    float *__restrict__ grads_sdf,
    const float *__restrict__ counter)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_sites)
    {
        return;
    }
    
    grads_sdf[idx] = counter[idx] == 0.0f ? 0.0f : grads_sdf[idx] / counter[idx];

    return;
}



__global__ void render_no_grad_kernel(
    const size_t num_rays,
    const float inv_s,
    const float *__restrict__ sdf_seg,
    const float *__restrict__ color_samples,
    const int *__restrict__ offsets,
    float *__restrict__ color_out,
    float *__restrict__ mask_out)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rays)
    {
        return;
    }
    
    float Wtotal = 0.0f;
    float4 color = trace_ray(sdf_seg, color_samples, offsets, &Wtotal, inv_s, idx);

    color_out[3*idx] = color.x + color.w * BACK_R;
    color_out[3*idx + 1] = color.y + color.w * BACK_G;
    color_out[3*idx + 2] = color.z + color.w * BACK_B;

    mask_out[idx] = color.w;

    return;
}


/** CPU functions **/
/** CPU functions **/
/** CPU functions **/


// *************************
void render_cuda(
    size_t num_rays,
    float inv_s,
    float mask_reg,
    torch::Tensor sdf_seg,
    torch::Tensor neighbors,
    torch::Tensor weights_seg,
    torch::Tensor color_samples,
    torch::Tensor true_color,
    torch::Tensor mask,
    torch::Tensor cell_ids,
    torch::Tensor offsets,
    torch::Tensor grad_space,
    torch::Tensor rays,
    torch::Tensor grads_sdf,
    torch::Tensor grads_color,
    torch::Tensor grads_sdf_net,
    torch::Tensor counter,
    torch::Tensor color_loss,
    torch::Tensor mask_loss)
{
        const int threads = 512;
        const int blocks = (num_rays + threads - 1) / threads; // ceil for example 8192 + 255 / 256 = 32
        AT_DISPATCH_FLOATING_TYPES( color_samples.type(),"render_cuda", ([&] {  
            render_kernel CUDA_KERNEL(blocks,threads) (
                num_rays,
                inv_s,
                mask_reg,
                sdf_seg.data_ptr<float>(),
                neighbors.data_ptr<int>(),
                weights_seg.data_ptr<float>(),
                color_samples.data_ptr<float>(),
                true_color.data_ptr<float>(),
                mask.data_ptr<float>(),
                cell_ids.data_ptr<int>(),
                offsets.data_ptr<int>(),
                grad_space.data_ptr<float>(),
                rays.data_ptr<float>(),
                grads_sdf.data_ptr<float>(),
                grads_color.data_ptr<float>(),
                grads_sdf_net.data_ptr<float>(),
                counter.data_ptr<float>(),
                color_loss.data_ptr<float>(),
                mask_loss.data_ptr<float>());
        }));

        // Need normalization ??
        /*const int threads_n = 1024;
        const int blocks_n = (num_sites + threads_n - 1) / threads_n; 
        AT_DISPATCH_FLOATING_TYPES( grads_sdf.type(),"normalize_grads_kernel", ([&] {  
            normalize_grads_kernel CUDA_KERNEL(blocks_n,threads_n) (
                num_sites,
                grads_sdf.data_ptr<float>(),
                counter.data_ptr<float>());
        }));*/
    hipDeviceSynchronize();
}

void normalize_grads_cuda(
    size_t num_sites,
    torch::Tensor grads_sdf,
    torch::Tensor counter)
{
        const int threads = 1024;
        const int blocks = (num_sites + threads - 1) / threads; // ceil for example 8192 + 255 / 256 = 32
        AT_DISPATCH_FLOATING_TYPES( grads_sdf.type(),"normalize_grads_kernel", ([&] {  
            normalize_grads_kernel CUDA_KERNEL(blocks,threads) (
                num_sites,
                grads_sdf.data_ptr<float>(),
                counter.data_ptr<float>());
        }));
}

void render_no_sdf_cuda(
    size_t num_rays,
    float inv_s,
    float mask_reg,
    torch::Tensor sdf_seg,
    torch::Tensor neighbors,
    torch::Tensor weights_seg,
    torch::Tensor color_samples,
    torch::Tensor true_color,
    torch::Tensor mask,
    torch::Tensor cell_ids,
    torch::Tensor offsets,
    torch::Tensor grads_sdf,
    torch::Tensor grads_color,
    torch::Tensor color_loss,
    torch::Tensor mask_loss)
{
        const int threads = 512;
        const int blocks = (num_rays + threads - 1) / threads; // ceil for example 8192 + 255 / 256 = 32
        AT_DISPATCH_FLOATING_TYPES( color_samples.type(),"render_no_sdf_cuda", ([&] {  
            render_no_sdf_kernel CUDA_KERNEL(blocks,threads) (
                num_rays,
                inv_s,
                mask_reg,
                sdf_seg.data_ptr<float>(),
                neighbors.data_ptr<int>(),
                weights_seg.data_ptr<float>(),
                color_samples.data_ptr<float>(),
                true_color.data_ptr<float>(),
                mask.data_ptr<float>(),
                cell_ids.data_ptr<int>(),
                offsets.data_ptr<int>(),
                grads_sdf.data_ptr<float>(),
                grads_color.data_ptr<float>(),
                color_loss.data_ptr<float>(),
                mask_loss.data_ptr<float>());
        }));

        // Need normalization ??
}



void render_no_grad_cuda(
    size_t num_rays,
    float inv_s,
    torch::Tensor sdf_seg,
    torch::Tensor color_samples,
    torch::Tensor offsets,
    torch::Tensor color_out,
    torch::Tensor mask_out)
{
        const int threads = 1024;
        const int blocks = (num_rays + threads - 1) / threads; // ceil for example 8192 + 255 / 256 = 32
        AT_DISPATCH_FLOATING_TYPES( color_samples.type(),"render_no_grad_cuda", ([&] {  
            render_no_grad_kernel CUDA_KERNEL(blocks,threads) (
                num_rays,
                inv_s,
                sdf_seg.data_ptr<float>(),
                color_samples.data_ptr<float>(),
                offsets.data_ptr<int>(),
                color_out.data_ptr<float>(),
                mask_out.data_ptr<float>());
        }));

        // Need normalization ??
}